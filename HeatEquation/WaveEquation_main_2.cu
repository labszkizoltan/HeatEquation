#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime_api.h"

#include "src/window/window_context.h"
#include "src/controls/observer.h"
#include "src/rendering/coloured_mesh.h"
#include "src/rendering/simple_mesh.h"
#include "src/rendering/shader.h"
#include "src/rendering/shader_sources/ColouredShaderSources.h"
#include "src/rendering/shader_sources/SimpleShaderSources.h"
#include "src/rendering/grid_factory/CreateGridData.h"
#include "src/utilities/Matrix_3D.h"

#include <stdio.h>

#include <cuda_gl_interop.h> // this has to be included after some other headers, not sure which ones, havent tried all possibilities so I put this to the end, but at first this caused a compile error!!!

#include "GlobalVariables.h"

//#include "src/cuda_kernels/HeatEquationKernels.cuh"
#include "src/cuda_kernels/WaveEquationKernels.cuh"

void mouse_scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{
    Observer* obsPtr = (Observer*)glfwGetWindowUserPointer(window);
    if (yoffset > 0) { obsPtr->ZoomIn(1.1f); } // PARAMETER zoom multiplier
    else if (yoffset < 0) { obsPtr->ZoomOut(1.1f); }
}

void SetTimeSpeed(MyWindow& appWindow, float& timeSpeed)
{
    if (appWindow.IsKeyPressed(GLFW_KEY_SPACE)) { timeSpeed = 0.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_1)) { timeSpeed = 0.1f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_2)) { timeSpeed = 1.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_3)) { timeSpeed = 2.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_4)) { timeSpeed = 8.0f; }
}


int main()
{
    uint32_t gridSize = 100;
    uint32_t gridElements = gridSize * gridSize;
    float amplitude = 0.2f * (float)100;

    MyWindow appWindow(g_WindowWidth, g_WindowHeight, "WaveEquation"); std::cout << glfwGetError(NULL) << "\n";
    glfwSetWindowPos(appWindow.GetWindow(), 100, 200); std::cout << glfwGetError(NULL) << "\n";
    appWindow.SetMouseScrollCallback(mouse_scroll_callback); std::cout << glfwGetError(NULL) << "\n";

    Observer observer;
    observer.translation = Vec3D(0.5f * (float)gridSize, 10.0f, -20.0f);// observer.TurnDown(0.5f);
    appWindow.SetUserPointer(&observer);

    // Create the grids
    //std::vector<Vec3D> vertexData = GridFactory::CreateGridVertexData(gridSize, amplitude);

    std::vector<float> g_WaveEquationInitialCondition_100x100;
    g_WaveEquationInitialCondition_100x100.resize(100 * 100);
    for (int i = 0; i < 100; i++)
    {
        for (int j = 0; j < 100; j++)
        {
            if (((i-25)*(i-25)+(j-25)*(j-25)) < 25)
                g_WaveEquationInitialCondition_100x100[100*i+j] = cos(0.63661977f/10.0f*(float)((i-25)*(i-25)+(j-25)*(j-25)));
        }
    }
//    g_WaveEquationInitialCondition_100x100[5050] = 1.0f;

    std::vector<float> flatGrid = GridFactory::MapAmplitudeFields(gridSize, g_FlatField_1x1);
    std::vector<float> initialConditionGrid = GridFactory::MapAmplitudeFields(gridSize, g_WaveEquationInitialCondition_100x100);

    std::vector<Vec3D> flatVertexData = GridFactory::CreateGridVertexData_with_amplitudes(gridSize, amplitude, flatGrid);
    std::vector<Vec3D> initialConditionVertexData = GridFactory::CreateGridVertexData_with_amplitudes(gridSize, amplitude, initialConditionGrid);
    std::vector<uint32_t> indexData = GridFactory::CreateGridIndexData(gridSize);
    SimpleMesh Grid_displacement(initialConditionVertexData, indexData);
    SimpleMesh Grid_velocity(flatVertexData, indexData);
    SimpleMesh Grid_acceleration(flatVertexData, indexData);

    Shader simpleShader(VertexShader_Simple, FragmentShader_Simple);
    {
        simpleShader.Bind();
        simpleShader.UploadUniformFloat3("body_translation", glm::vec3(0.0f, 0.0f, 0.0f));
        simpleShader.UploadUniformMat3("body_orientation", glm::mat3(1.0f));
        simpleShader.UploadUniformFloat("body_scale", 1.0f);
        simpleShader.UploadUniformFloat3("observer_translation", glm::vec3(0.0f, 5.0f, -10.0f));
        simpleShader.UploadUniformMat3("observer_orientation", glm::mat3(1.0f));
        simpleShader.UploadUniformFloat("zoom_level", 1.0f);
        simpleShader.UploadUniformFloat("aspect_ratio", (float)g_WindowWidth / (float)g_WindowHeight);
        simpleShader.UploadUniformFloat("amplitude", amplitude/10.0f);
    }


    struct hipGraphicsResource* cuda_vbo_resource_1;
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource_1, (GLuint)Grid_displacement.m_VertexBuffer.m_RendererID, cudaGraphicsMapFlagsNone));
    struct hipGraphicsResource* cuda_vbo_resource_2;
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource_2, (GLuint)Grid_velocity.m_VertexBuffer.m_RendererID, cudaGraphicsMapFlagsNone));
    struct hipGraphicsResource* cuda_vbo_resource_3;
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource_3, (GLuint)Grid_acceleration.m_VertexBuffer.m_RendererID, cudaGraphicsMapFlagsNone));

    uint32_t blockSize = 320;

    float timeSpeed = 0.0f; // PARAMETER initial time speed
    float timestep = 0.005f; // timestep can be initialized like this, because its constructor takes in only one float, implicit cast is possible

    int counter = 0, draw_frequency = 10;

    // Game loop
    while (!glfwWindowShouldClose(appWindow.GetWindow()))
    {
//        appWindow.HandleUserInputs(observer, timestep);

        // Set the speed of the simulation, note that the quality of the update will be worse, as the timestep will be bigger
        SetTimeSpeed(appWindow, timeSpeed);

//        observer.SetObserverInShader(simpleShader);

        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);


        if (timeSpeed > 0.0f)
        {

            // --------------------- //
            // do the CUDA part here //

            float3* dptr_r, * dptr_v, * dptr_a;
            size_t num_bytes;
            checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource_1, 0));
            checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource_2, 0));
            checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource_3, 0));
            checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr_r, &num_bytes, cuda_vbo_resource_1));
            checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr_v, &num_bytes, cuda_vbo_resource_2));
            checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr_a, &num_bytes, cuda_vbo_resource_3));


            __global__ void UpdateDisplacement_kernel(float3 * displacement, float3 * velocity, unsigned int gridSize, float deltaTime);
            __global__ void UpdateVelocity_kernel(float3 * velocity, float3 * acceleration, unsigned int gridSize, float deltaTime);
            __global__ void UpdateAcceleration_kernel(float3 * acceleration, float3 * displacement, unsigned int gridSize, float deltaTime);

            // launch kernels here
            UpdateDisplacement_kernel <<<gridElements / blockSize + 1, blockSize >>> (dptr_r, dptr_v, gridSize, timestep*timeSpeed);
            UpdateVelocity_kernel     <<<gridElements / blockSize + 1, blockSize >>> (dptr_v, dptr_a, gridSize, timestep*timeSpeed);
            UpdateAcceleration_kernel <<<gridElements / blockSize + 1, blockSize >>> (dptr_a, dptr_r, gridSize, timestep*timeSpeed);

            checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource_1, 0));
            checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource_2, 0));
            checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource_3, 0));

            // --------------------- //

        }

        if (counter > draw_frequency)
        {
            appWindow.HandleUserInputs(observer, timestep);
            observer.SetObserverInShader(simpleShader);
            Grid_displacement.Draw();
            glfwSwapBuffers(appWindow.GetWindow());
            counter = 0;
        }
        counter++;
        // Swap the screen buffers
//        glfwSwapBuffers(appWindow.GetWindow());
    }

    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource_1));
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource_2));
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource_3));

    // Terminates GLFW, clearing any resources allocated by GLFW.
    glfwTerminate();

    return 0;
}


