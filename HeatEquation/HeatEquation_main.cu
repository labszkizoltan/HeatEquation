#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include "hip/hip_runtime_api.h"

#include "src/window/window_context.h"
#include "src/controls/observer.h"
#include "src/rendering/coloured_mesh.h"
#include "src/rendering/simple_mesh.h"
#include "src/rendering/shader.h"
#include "src/rendering/shader_sources/ColouredShaderSources.h"
#include "src/rendering/shader_sources/SimpleShaderSources.h"
#include "src/rendering/grid_factory/CreateGridData.h"

#include "src/utilities/Matrix_3D.h"


#include <stdio.h>

#include <cuda_gl_interop.h> // this has to be included after some other headers, not sure which ones, havent tried all possibilities so I put this to the end, but at first this caused a compile error!!!

const int windowWidth = 1200;
const int windowHeight = 800;

void mouse_scroll_callback(GLFWwindow* window, double xoffset, double yoffset)
{
    Observer* obsPtr = (Observer*)glfwGetWindowUserPointer(window);
    if (yoffset > 0) { obsPtr->ZoomIn(1.1f); } // PARAMETER zoom multiplier
    else if (yoffset < 0) { obsPtr->ZoomOut(1.1f); }
}

void SetTimeSpeed(MyWindow& appWindow, float& timeSpeed)
{
    if (appWindow.IsKeyPressed(GLFW_KEY_SPACE)) { timeSpeed = 0.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_1)) { timeSpeed = 1.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_2)) { timeSpeed = 2.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_3)) { timeSpeed = 8.0f; }
    if (appWindow.IsKeyPressed(GLFW_KEY_4)) { timeSpeed = 16.0f; }
}


__global__ void simple_vbo_kernel(float3* pos_1, float3* pos_2, unsigned int gridSize, float time)
{
//    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
//    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    unsigned int x = threadIdx.x;
    unsigned int y = threadIdx.y;

    // prime numbers from https://primes.utm.edu/lists/small/10000.txt
    int quasiRand = (101119*(threadIdx.x+threadIdx.y+(int)time)+82031)%7993;
    float qRand = (float)quasiRand / (float)7993 - 0.5f;


    // write output vertex
    pos_1[y * gridSize + x].y += 0.5f*qRand;
    pos_1[y * gridSize + x].y *= 0.997f;
}


__global__ void HeatEquation_kernel(float3* target, float3* source, unsigned int gridSize, float deltaTime)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;

//    if (i < (gridSize * gridSize-1) && i>1)
    if (i < (gridSize*gridSize))
    {
        unsigned int x = i / gridSize;
        unsigned int y = i % gridSize;

        unsigned int x_next = (x == (gridSize - 1)) ? 0 : x + 1;
        unsigned int x_prev = (x == 0) ? (gridSize - 1) : x - 1;
        unsigned int y_next = (y == (gridSize - 1)) ? 0 : y + 1;
        unsigned int y_prev = (y == 0) ? (gridSize - 1) : y - 1;

        // write output vertex
        target[y * gridSize + x].y = source[y * gridSize + x].y + deltaTime * (source[y_next * gridSize + x].y + source[y_prev * gridSize + x].y + source[y * gridSize + x_next].y + source[y * gridSize + x_prev].y - 4.0f * source[y * gridSize + x].y);
    }
}


__global__ void SyncVertexBuffers_kernel(float3* target, float3* source, unsigned int gridSize)
{
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < (gridSize * gridSize))
    {
        target[i].y = source[i].y;
    }
}





int main()
{
    uint32_t gridSize = 1000;
    uint32_t gridElements = gridSize * gridSize;
    float amplitude = 500.0f;

    // CUDA part over, lets try setting up a window

    MyWindow appWindow(windowWidth, windowHeight, "HeatEquation"); std::cout << glfwGetError(NULL) << "\n";
    glfwSetWindowPos(appWindow.GetWindow(), 100, 200); std::cout << glfwGetError(NULL) << "\n";
    appWindow.SetMouseScrollCallback(mouse_scroll_callback); std::cout << glfwGetError(NULL) << "\n";

    Observer observer;
    observer.translation = Vec3D(0.5f * (float)gridSize, 10.0f, -20.0f);// observer.TurnDown(0.5f);
    appWindow.SetUserPointer(&observer);

    // create a simple non coloured mesh, just a triangle
    std::vector<Vec3D> vertexData = GridFactory::CreateGridVertexData(gridSize, amplitude);
    std::vector<uint32_t> indexData = GridFactory::CreateGridIndexData(gridSize);
    SimpleMesh GridMesh_1(vertexData, indexData);
    SimpleMesh GridMesh_2(vertexData, indexData);

    Shader simpleShader(VertexShader_Simple, FragmentShader_Simple);
    {
        simpleShader.Bind();
        simpleShader.UploadUniformFloat3("body_translation", glm::vec3(0.0f, 0.0f, 0.0f));
        simpleShader.UploadUniformMat3("body_orientation", glm::mat3(1.0f));
        simpleShader.UploadUniformFloat("body_scale", 1.0f);
        simpleShader.UploadUniformFloat3("observer_translation", glm::vec3(0.0f, 5.0f, -10.0f));
        simpleShader.UploadUniformMat3("observer_orientation", glm::mat3(1.0f));
        simpleShader.UploadUniformFloat("zoom_level", 1.0f);
        simpleShader.UploadUniformFloat("aspect_ratio", (float)windowWidth / (float)windowHeight);
        simpleShader.UploadUniformFloat("amplitude", amplitude);
    }


    // Cuda functions that need to be called
//  hipGraphicsGLRegisterBuffer // once after the vertex buffer has been created
//  hipGraphicsMapResources // every time in the rendering loop
//  hipGraphicsResourceGetMappedPointer // every time in the rendering loop
//  hipGraphicsUnmapResources // every time in the rendering loop
//  hipGraphicsUnregisterResource // once after the vertex buffer has been destroyed


    struct hipGraphicsResource* cuda_vbo_resource_1;
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource_1, (GLuint)GridMesh_1.m_VertexBuffer.m_RendererID, cudaGraphicsMapFlagsNone));
    struct hipGraphicsResource* cuda_vbo_resource_2;
    checkCudaErrors(hipGraphicsGLRegisterBuffer(&cuda_vbo_resource_2, (GLuint)GridMesh_2.m_VertexBuffer.m_RendererID, cudaGraphicsMapFlagsNone));


    float time = (float)glfwGetTime();
    float timeSpeed = 0.0f; // PARAMETER initial time speed
    float timestep = 0.0f; // timestep can be initialized like this, because its constructor takes in only one float, implicit cast is possible
    float lastFrameTime = 0.0f;

    // Game loop
    while (!glfwWindowShouldClose(appWindow.GetWindow()))
    {
        lastFrameTime = (float)glfwGetTime();
//        appWindow.HandleUserInputs(observer, timestep*timeSpeed);
        appWindow.HandleUserInputs(observer, timestep);

        // Set the speed of the simulation, note that the quality of the update will be worse, as the timestep will be bigger
        SetTimeSpeed(appWindow, timeSpeed);

        observer.SetObserverInShader(simpleShader);
        
        glClearColor(0.0f, 0.0f, 0.0f, 1.0f);
        glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

        // --------------------- //
        // do the CUDA part here //

        float3 *dptr_1, *dptr_2;
        size_t num_bytes;

        checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource_1, 0));
        checkCudaErrors(hipGraphicsMapResources(1, &cuda_vbo_resource_2, 0));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr_1, &num_bytes, cuda_vbo_resource_1));
        checkCudaErrors(hipGraphicsResourceGetMappedPointer((void**)&dptr_2, &num_bytes, cuda_vbo_resource_1));

        // launch kernel here
        HeatEquation_kernel <<<gridElements/256+1, 256>>> (dptr_1, dptr_2, gridSize, timestep*timeSpeed);
//        checkCudaErrors(hipDeviceSynchronize());
        HeatEquation_kernel <<<gridElements/256+1, 256>>> (dptr_2, dptr_1, gridSize, timestep*timeSpeed);

        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource_1, 0));
        checkCudaErrors(hipGraphicsUnmapResources(1, &cuda_vbo_resource_2, 0));

        // --------------------- //


        GridMesh_1.Draw();

        // Swap the screen buffers
        glfwSwapBuffers(appWindow.GetWindow());
        std::cout << glfwGetError(NULL) << "\n";

        timestep = (float)glfwGetTime() - lastFrameTime;
        //		timestep = 0.017f;
    }

    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource_1));
    checkCudaErrors(hipGraphicsUnregisterResource(cuda_vbo_resource_2));

    // Terminates GLFW, clearing any resources allocated by GLFW.
    glfwTerminate();

    return 0;
}


